
#include "hip/hip_runtime.h"




#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define CUDA_CALLABLE_MEMBER
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#include <stdio.h>

const int N = 16;
const int blocksize = 16;

__global__ void hello(char* a, int* b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = { 15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	char* ad;
	int* bd;
	const int csize = N * sizeof(char);
	const int isize = N * sizeof(int);

	printf("%s", a);

	hipMalloc((void**)& ad, csize);
	hipMalloc((void**)& bd, isize);
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	hello KERNEL_ARGS2(dimGrid, dimBlock) (ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(bd);

	printf("%s\n", a);
	return EXIT_SUCCESS;
}